#include "hip/hip_runtime.h"
#include "helpers.h"
#include "constants.h"
#include "cuda_helpers.cuh"
#include "kernelfunctions.cuh"
#include <string>
#include <iostream>

int main(int argc, char *argv[])
{
    // Checking Command Line

    int n_params = 7;       // Expected number of params
    std::string usage = ""; // Usage description

    std::string example = ""; // Example usage

    // Check Number of CMD Line arguments
    g3lcong::checkCmdLine(argc, n_params, usage, example);

    // Read in command line arguments
    std::string filename_sources = argv[1]; // File with Source Galaxies
    std::string filename_lenses1 = argv[2]; // File with Lens1 Galaxies
    std::string filename_lenses2 = argv[3]; // File with Lens2 Galaxies

    double theta_min = std::stod(argv[4]); // Min Theta for calc of Gtilde [arcmin]
    double theta_max = std::stod(argv[5]); // Max Theta for calc of Gtilde [arcmin]
    int num_bins = std::stoi(argv[6]);     // Number of Bins for Gtilde on ONE axis

    double deltaChi = std::stod(argv[7]);

    double phi_min = 0; // Min Phi for calc of Gtilde [radians]
    double phi_max = 2*g3lcong::pi;  // Max Phi for calc of Gtilde [radians]

    // Reading in galaxies and copying to device

    // x,y, z vectors
    std::vector<double> x1, y1, z1, chi1, x2, y2, z2, chi2, xS, yS, e1, e2, w, chiS;

    if (g3lcong::readPhysSources(filename_sources, 7, 1, 2, 3, 4, 6, 7, xS, yS, e1, e2, w, chiS))
        return 1;
    if (g3lcong::readPhysLenses(filename_lenses1, 7, 1, 2, 7, x1, y1, chi1))
        return 1;
    if (g3lcong::readPhysLenses(filename_lenses2, 7, 1, 2, 7, x2, y2, chi2))
        return 1;

    // Declare arrays for coordinates of galaxies on device
    double *dev_x1, *dev_y1, *dev_x2, *dev_y2, *dev_chi1, *dev_chi2;
    double *dev_xS, *dev_yS, *dev_e1, *dev_e2, *dev_w, *dev_chiS;

    // Numbers of sources and lenses
    int N1, N2, NS;
    N1 = x1.size(); // Number of galaxies
    N2 = x2.size();
    NS = xS.size();

    // Allocate memory on device
    CUDA_SAFE_CALL(hipMalloc(&dev_x1, N1 * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&dev_y1, N1 * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&dev_chi1, N1 * sizeof(double)));

    CUDA_SAFE_CALL(hipMalloc(&dev_x2, N2 * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&dev_y2, N2 * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&dev_chi2, N2 * sizeof(double)));

    CUDA_SAFE_CALL(hipMalloc(&dev_xS, NS * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&dev_yS, NS * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&dev_e1, NS * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&dev_e2, NS * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&dev_w, NS * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&dev_chiS, NS * sizeof(double)));

    // Copy values
    CUDA_SAFE_CALL(hipMemcpy(dev_x1, x1.data(), N1 * sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(dev_y1, y1.data(), N1 * sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(dev_chi1, chi1.data(), N1 * sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(dev_x2, x2.data(), N2 * sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(dev_y2, y2.data(), N2 * sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(dev_chi2, chi2.data(), N2 * sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(dev_xS, xS.data(), NS * sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(dev_yS, yS.data(), NS * sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(dev_e1, e1.data(), NS * sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(dev_e2, e2.data(), NS * sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(dev_w, w.data(), NS * sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(dev_chiS, chiS.data(), NS * sizeof(double), hipMemcpyHostToDevice));

    // Calculate Gtilde

    // Declare container for Greal, Gimag and weight on device
    double *dev_Greal, *dev_Gimag, *dev_weight;

    // Allocate memory for Greal, Gimag and weight on device
    CUDA_SAFE_CALL(hipMalloc(&dev_Greal, num_bins * num_bins * num_bins * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&dev_Gimag, num_bins * num_bins * num_bins * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&dev_weight, num_bins * num_bins * num_bins * sizeof(double)));

    // Check if memory for Gtilde could be allocated on device
    if (0 == dev_Greal || 0 == dev_Gimag || 0 == dev_weight)
    {
        std::cout << "Couldnt allocate Space for Gtilde" << std::endl;
        return 1;
    };

    // Set Gtilde to 0 on device
    CUDA_SAFE_CALL(hipMemset(dev_Greal, 0, num_bins * num_bins * num_bins * sizeof(double)));
    CUDA_SAFE_CALL(hipMemset(dev_Gimag, 0, num_bins * num_bins * num_bins * sizeof(double)));
    CUDA_SAFE_CALL(hipMemset(dev_weight, 0, num_bins * num_bins * num_bins * sizeof(double)));

    g3lcong::addToGGE<<<BLOCKS, THREADS>>>(dev_x1, dev_y1,
                                           dev_x2, dev_y2,
                                           dev_xS, dev_yS,
                                           dev_chi1, dev_chi2, dev_chiS,
                                           dev_e1, dev_e2,
                                           dev_w, deltaChi,
                                           num_bins, N1, N2, NS,
                                           theta_min, theta_max,
                                           dev_Greal,
                                           dev_Gimag, dev_weight);

    // Declare arrays for total Greal, Gimag and weight on host
    double *Greal_tot, *Gimag_tot, *weight_tot;

    // Allocate memory for total Greal, Gimag and weight on host
    Greal_tot = (double *)malloc(num_bins * num_bins * num_bins * sizeof(double));
    Gimag_tot = (double *)malloc(num_bins * num_bins * num_bins * sizeof(double));
    weight_tot = (double *)malloc(num_bins * num_bins * num_bins * sizeof(double));

    if (Greal_tot == NULL || Gimag_tot == NULL || weight_tot == NULL)
    {
        std::cerr << "calculateGtilde_gpu: Couldn't allocate memory" << std::endl;
        exit(1);
    };

    // Copy Gtilde from device to host
    CUDA_SAFE_CALL(hipMemcpy(Greal_tot, dev_Greal, num_bins * num_bins * num_bins * sizeof(double),
                              hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(Gimag_tot, dev_Gimag, num_bins * num_bins * num_bins * sizeof(double),
                              hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(weight_tot, dev_weight, num_bins * num_bins * num_bins * sizeof(double),
                              hipMemcpyDeviceToHost));

    // Output

    // Print out vartheta1, vartheta2, psi, binsize, Gtilde
    double phi_binsize = (phi_max - phi_min) / num_bins;
    double theta_binsize = log(theta_max / theta_min) / num_bins;

    for (int i = 0; i < num_bins; i++)
    {
        // Theta1 Center of this bin
        double theta = 0.5 * (exp(log(theta_min) + theta_binsize * (i + 1)) + exp(log(theta_min) + theta_binsize * i));
        // Theta1 Binsize of this bin
        double deltaTheta = exp(log(theta_min) + theta_binsize * (i + 1)) - exp(log(theta_min) + theta_binsize * i);

        for (int j = 0; j < num_bins; j++)
        {
            double phi1 = (j + 0.5) * phi_binsize + phi_min;

            for (int k = 0; k < num_bins; k++)
            {
                // Phi Center of this bin
                double phi2 = (k + 0.5) * phi_binsize + phi_min;

                int index = i * num_bins * num_bins + j * num_bins + k;

                // Weight
                double weight = weight_tot[index];
                // Greal
                double Greal = Greal_tot[index];
                // Gimag
                double Gimag = Gimag_tot[index];

                if (weight != 0)
                {
                    Greal /= weight;
                    Gimag /= weight;
                }
                else
                {
                    Greal = 0;
                    Gimag = 0;
                }

                // Output
                std::cout
                    << theta << " "       // bin center theta 1 [Mpc/h]
                    << phi1 << " "        // bin center theta 2 [Mpc/h]
                    << phi2 << " "        // phi center [radians]
                    << deltaTheta << " "  // bin size theta 1[arcmin or Mpc]
                    << phi_binsize << " " // bin size theta 2[arcmin or Mpc]
                    << phi_binsize << " " // phi bin size [radians]
                    << Greal << " "       // Real part of Gtilde [dimensionless or Msun/Mpc²]
                    << Gimag << " "       // Imaginary part of Gtilde [dimensionless or Msun/Mpc²]
                    << weight             // Weight of Gtilde [dimensionless or Msun/Mpc²]
                    << std::endl;
            };
        };
    };

    return 0;
}